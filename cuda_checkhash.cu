#include "hip/hip_runtime.h"
/**
 * This code compares final hash against target
 */
#include <stdio.h>
#include <memory.h>

#include "miner.h"
#include "cuda_helper.h"

__constant__ uint32_t pTarget[8]; // 32 bytes

// store MAX_GPUS device arrays of 8 nonces
static uint32_t* h_resNonces[MAX_GPUS];
static uint32_t* d_resNonces[MAX_GPUS];

__host__
void cuda_check_cpu_init(int thr_id, uint32_t threads)
{
    CUDA_SAFE_CALL(hipHostMalloc(&h_resNonces[thr_id], 8*sizeof(uint32_t)));
	CUDA_SAFE_CALL(hipMalloc(&d_resNonces[thr_id], 8 * sizeof(uint32_t)));
}

// Target Difficulty

__host__
void cuda_check_cpu_setTarget(const void *ptarget, int thr_id)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(pTarget), ptarget, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice, gpustream[thr_id]));
}

/* --------------------------------------------------------------------------------------------- */

__device__ __forceinline__
static bool hashbelowtarget(const uint32_t *const __restrict__ hash, const uint32_t *const __restrict__ target)
{
	if (hash[7] > target[7])
		return false;
	if (hash[7] < target[7])
		return true;
	if (hash[6] > target[6])
		return false;
	if (hash[6] < target[6])
		return true;

	if (hash[5] > target[5])
		return false;
	if (hash[5] < target[5])
		return true;
	if (hash[4] > target[4])
		return false;
	if (hash[4] < target[4])
		return true;

	if (hash[3] > target[3])
		return false;
	if (hash[3] < target[3])
		return true;
	if (hash[2] > target[2])
		return false;
	if (hash[2] < target[2])
		return true;

	if (hash[1] > target[1])
		return false;
	if (hash[1] < target[1])
		return true;
	if (hash[0] > target[0])
		return false;

	return true;
}

__global__ __launch_bounds__(512, 2)
void cuda_checkhash_64(uint32_t threads, uint32_t startNounce, uint32_t *hash, uint32_t *resNonces)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// shl 4 = *16 x 4 (uint32) = 64 bytes
		// todo: use only 32 bytes * threads if possible
		uint32_t *inpHash = &hash[thread << 4];

		if (resNonces[0] == UINT32_MAX) {
			if (hashbelowtarget(inpHash, pTarget))
				resNonces[0] = (startNounce + thread);
		}
	}
}

__host__
uint32_t cuda_check_hash(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_inputHash)
{
	CUDA_SAFE_CALL(hipMemsetAsync(d_resNonces[thr_id], 0xff, sizeof(uint32_t), gpustream[thr_id]));

	const uint32_t threadsperblock = 512;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	cuda_checkhash_64 <<<grid, block, 0, gpustream[thr_id]>>> (threads, startNounce, d_inputHash, d_resNonces[thr_id]);

	hipMemcpyAsync(h_resNonces[thr_id], d_resNonces[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost, gpustream[thr_id]);
	hipStreamSynchronize(gpustream[thr_id]);

	return h_resNonces[thr_id][0];
}

/* --------------------------------------------------------------------------------------------- */

__global__ __launch_bounds__(512, 2)
void cuda_checkhash_64_suppl(uint32_t startNounce, uint32_t *hash, uint32_t *resNonces)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	uint32_t *inpHash = &hash[thread << 4];

	if (hashbelowtarget(inpHash, pTarget)) {
		int resNum = atomicAdd(resNonces,1)+1;
		if (resNum < 8)
			resNonces[resNum] = (startNounce + thread);
	}
}

__host__
uint32_t cuda_check_hash_suppl(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_inputHash, uint32_t foundnonce)
{
	uint32_t rescnt, result = 0;

	const uint32_t threadsperblock = 512;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	// first element stores the count of found nonces
	hipMemsetAsync(d_resNonces[thr_id], 0, sizeof(uint32_t), gpustream[thr_id]);

	cuda_checkhash_64_suppl <<<grid, block, 0, gpustream[thr_id]>>> (startNounce, d_inputHash, d_resNonces[thr_id]);
	hipMemcpyAsync(h_resNonces[thr_id], d_resNonces[thr_id], 8*sizeof(uint32_t), hipMemcpyDeviceToHost, gpustream[thr_id]);
	hipStreamSynchronize(gpustream[thr_id]);

	rescnt = h_resNonces[thr_id][0];
	if (rescnt > 1)
	{
		do
		{
			if (h_resNonces[thr_id][rescnt] != foundnonce)
			{
				result = h_resNonces[thr_id][rescnt];
				break;
			}
			rescnt--;
		} while (rescnt > 0);
	}
	return result;
}

/* --------------------------------------------------------------------------------------------- */

__global__
void cuda_check_hash_branch_64(uint32_t threads, uint32_t startNounce, uint32_t *g_nonceVector, uint32_t *g_hash, uint32_t *resNounce)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nounce = g_nonceVector[thread];
		uint32_t hashPosition = (nounce - startNounce) << 4;
		const uint32_t *const inpHash = &g_hash[hashPosition];

		if (hashbelowtarget(inpHash, pTarget))
		{
			if (resNounce[0] > nounce)
				resNounce[0] = nounce;
		}
	}
}

__global__
void cuda_check_quarkcoin_64(uint32_t threads, uint32_t startNounce, uint32_t *g_nonceVector, uint32_t *g_hash, uint32_t *resNounce)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nounce = g_nonceVector[thread];
		uint32_t hashPosition = (nounce - startNounce) << 4;
		const uint32_t *const inpHash = &g_hash[hashPosition];

		if (inpHash[7] <= pTarget[7])
		{
			uint32_t tmp = atomicExch(resNounce, nounce);
			if (tmp != 0xffffffff)
				resNounce[1] = tmp;
		}
	}
}

__host__
uint32_t cuda_check_hash_branch(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash)
{
	uint32_t result = 0xffffffff;
	hipMemsetAsync(d_resNonces[thr_id], 0xff, sizeof(uint32_t), gpustream[thr_id]);

	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	cuda_check_hash_branch_64 <<<grid, block, 0, gpustream[thr_id]>>> (threads, startNounce, d_nonceVector, d_inputHash, d_resNonces[thr_id]);

	hipMemcpyAsync(h_resNonces[thr_id], d_resNonces[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost, gpustream[thr_id]);
	hipStreamSynchronize(gpustream[thr_id]);

	result = *h_resNonces[thr_id];

	return result;
}
__host__
void cuda_check_quarkcoin(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash, uint32_t *resNonces)
{
	CUDA_SAFE_CALL(hipMemsetAsync(d_resNonces[thr_id], 0xff, 2 * sizeof(uint32_t), gpustream[thr_id]));

	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	cuda_check_quarkcoin_64 << <grid, block, 0, gpustream[thr_id]>>> (threads, startNounce, d_nonceVector, d_inputHash, d_resNonces[thr_id]);

	hipMemcpyAsync(resNonces, d_resNonces[thr_id], 2*sizeof(uint32_t), hipMemcpyDeviceToHost, gpustream[thr_id]);
	hipStreamSynchronize(gpustream[thr_id]);
}

int cuda_arch[MAX_GPUS];
__global__ void get_cuda_arch_gpu(int *d_version)
{
#ifdef __CUDA_ARCH__
	*d_version = __CUDA_ARCH__;
#endif
}

extern sha_algos opt_algo;

__host__ void get_cuda_arch(int *version)
{
	int *d_version;
	hipMalloc(&d_version, sizeof(int));
	get_cuda_arch_gpu << < 1, 1 >> > (d_version);
	hipMemcpy(version, d_version, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_version);
}
